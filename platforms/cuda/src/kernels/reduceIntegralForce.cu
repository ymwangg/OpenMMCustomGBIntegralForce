#include "hip/hip_runtime.h"
extern "C" __global__ void reduceGBSWForce(unsigned long long* __restrict__ forceBuffers,
        const real4* __restrict__ posq 
#ifdef USE_PERIODIC
        ,const real4 periodicBoxSize, const real4 invPeriodicBoxSize
#endif
PARAM_ARGS)
{   
    float3 minCoor = make_float3(lookupTableMinCoor[0],lookupTableMinCoor[1],lookupTableMinCoor[2]);
    int3 numGridPoints = make_int3(lookupTableNumGridPoints[0],lookupTableNumGridPoints[1],lookupTableNumGridPoints[2]);
    float3 gridStep = make_float3(lookupTableGridStep[0], lookupTableGridStep[1], lookupTableGridStep[2]);

    INIT_PARAMS

    //each blockIdx.x maps to an atom
    for(int atomI = blockIdx.x; atomI < NUM_ATOMS; atomI += gridDim.x){
        float3 forceI = make_float3(0,0,0);
        float4 posI = posq[atomI];
        int quadIdx = blockIdx.y*blockDim.x + threadIdx.x;
        LOAD_VOLUME
        if(quadIdx < NUM_QUADRATURE_POINTS){
        float4 quadPosR = make_float4(posI.x + quad[quadIdx].x, 
                posI.y + quad[quadIdx].y, posI.z + quad[quadIdx].z, quad[quadIdx].w);
#ifdef USE_PERIODIC
        float3 quadPos0 = make_float3(
                quadPosR.x - periodicBoxSize.x * floorf((quadPosR.x - minCoor.x)/periodicBoxSize.x),
                quadPosR.y - periodicBoxSize.y * floorf((quadPosR.y - minCoor.y)/periodicBoxSize.y),
                quadPosR.z - periodicBoxSize.z * floorf((quadPosR.z - minCoor.z)/periodicBoxSize.z));
        int3 lookupTableIdx3 = make_int3(
                (quadPos0.x - minCoor.x)/gridStep.x,
                (quadPos0.y - minCoor.y)/gridStep.y,
                (quadPos0.z - minCoor.z)/gridStep.z);
#else
        int3 lookupTableIdx3 = make_int3(
                (quadPosR.x - minCoor.x)/gridStep.x,
                (quadPosR.y - minCoor.y)/gridStep.y,
                (quadPosR.z - minCoor.z)/gridStep.z);
#endif
        BEFORE_VOLUME
#ifdef USE_LOOKUP_TABLE
        int lookupTableIdx = (lookupTableIdx3.x*(numGridPoints.y*numGridPoints.z) + 
                lookupTableIdx3.y*numGridPoints.z + lookupTableIdx3.z);
        int numLookupTableAtoms = lookupTableNumAtoms[lookupTableIdx];
        for(int i = 0; i<numLookupTableAtoms; i++){
            int atomJ = lookupTable[lookupTableIdx*LOOKUPTABLE_SIZE + i]; 
#else
        for(int atomJ = 0; atomJ < NUM_ATOMS; atomJ++){
#endif
            float3 forceJ = make_float3(0,0,0);
            float4 posJ = posq[atomJ];
            float3 delta = make_float3(posJ.x - quadPosR.x, 
                    posJ.y - quadPosR.y, posJ.z - quadPosR.z);
#ifdef USE_PERIODIC
            APPLY_PERIODIC_TO_DELTA(delta);
#endif
            COMPUTE_FORCE
            APPLY_CHAIN_RULE

            atomicAdd(&forceBuffers[atomJ], (long long) (forceJ.x*0x100000000));
            atomicAdd(&forceBuffers[atomJ+NUM_PADDED_ATOMS], (long long) (forceJ.y*0x100000000));
            atomicAdd(&forceBuffers[atomJ+NUM_PADDED_ATOMS*2], (long long) (forceJ.z*0x100000000));
        } // end atomJ
        atomicAdd(&forceBuffers[atomI], (long long) (forceI.x*0x100000000));
        atomicAdd(&forceBuffers[atomI+NUM_PADDED_ATOMS], (long long) (forceI.y*0x100000000));
        atomicAdd(&forceBuffers[atomI+NUM_PADDED_ATOMS*2], (long long) (forceI.z*0x100000000));
        } // end quadrature point
    } // end atomI
}
