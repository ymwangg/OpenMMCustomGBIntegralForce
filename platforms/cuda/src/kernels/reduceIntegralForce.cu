#include "hip/hip_runtime.h"
extern "C" __global__ void reduceGBSWForce(unsigned long long* __restrict__ forceBuffers,
        const real4* __restrict__ posq 
#ifdef USE_PERIODIC
        ,const real4 periodicBoxSize, const real4 invPeriodicBoxSize
#endif
PARAM_ARGS)
{   
    float3 minCoor = make_float3(lookupTableMinCoor[0],lookupTableMinCoor[1],lookupTableMinCoor[2]);
    int3 numGridPoints = make_int3(lookupTableNumGridPoints[0],lookupTableNumGridPoints[1],lookupTableNumGridPoints[2]);
    float3 gridStep = make_float3(lookupTableGridStep[0], lookupTableGridStep[1], lookupTableGridStep[2]);

    INIT_PARAMS

    //each blockIdx.x maps to an atom
    for(int atomI = blockIdx.x; atomI < NUM_ATOMS; atomI += gridDim.x){
        float3 forceI = make_float3(0.0,0.0,0.0);
        float4 posI = posq[atomI];
        int quadIdx = blockIdx.y*blockDim.x + threadIdx.x;
        LOAD_VOLUME
        if(quadIdx < NUM_QUADRATURE_POINTS){
        float4 quadPosR = make_float4(posI.x + quad[quadIdx].x, 
                posI.y + quad[quadIdx].y, posI.z + quad[quadIdx].z, quad[quadIdx].w);
#ifdef USE_PERIODIC
        float3 quadPos0 = make_float3(
                quadPosR.x - periodicBoxSize.x * floorf((quadPosR.x - minCoor.x)/periodicBoxSize.x),
                quadPosR.y - periodicBoxSize.y * floorf((quadPosR.y - minCoor.y)/periodicBoxSize.y),
                quadPosR.z - periodicBoxSize.z * floorf((quadPosR.z - minCoor.z)/periodicBoxSize.z));
        int3 lookupTableIdx3 = make_int3(
                (quadPos0.x - minCoor.x)/gridStep.x,
                (quadPos0.y - minCoor.y)/gridStep.y,
                (quadPos0.z - minCoor.z)/gridStep.z);
#else
        int3 lookupTableIdx3 = make_int3(
                (quadPosR.x - minCoor.x)/gridStep.x,
                (quadPosR.y - minCoor.y)/gridStep.y,
                (quadPosR.z - minCoor.z)/gridStep.z);
#endif
        BEFORE_VOLUME
#ifdef USE_LOOKUP_TABLE
        int lookupTableIdx = (lookupTableIdx3.x*(numGridPoints.y*numGridPoints.z) + 
                lookupTableIdx3.y*numGridPoints.z + lookupTableIdx3.z);
        int numLookupTableAtoms = lookupTableNumAtoms[lookupTableIdx];
        for(int i = 0; i<numLookupTableAtoms; i++){
            int atomJ = lookupTable[lookupTableIdx*LOOKUPTABLE_SIZE + i]; 
#else
        for(int atomJ = 0; atomJ < NUM_ATOMS; atomJ++){
#endif
            float3 forceJ = make_float3(0.0,0.0,0.0);
            float4 posJ = posq[atomJ];
            float3 delta = make_float3(posJ.x - quadPosR.x, 
                    posJ.y - quadPosR.y, posJ.z - quadPosR.z);
#ifdef USE_PERIODIC
            APPLY_PERIODIC_TO_DELTA(delta);
#endif
            COMPUTE_FORCE
            APPLY_CHAIN_RULE


            if(abs(forceJ.x)>1e-10) atomicAdd(&forceBuffers[atomJ], static_cast<unsigned long long>((long long) (forceJ.x*0x100000000)));
            if(abs(forceJ.y)>1e-10) atomicAdd(&forceBuffers[atomJ+NUM_PADDED_ATOMS], static_cast<unsigned long long>((long long) (forceJ.y*0x100000000)));
            if(abs(forceJ.z)>1e-10) atomicAdd(&forceBuffers[atomJ+NUM_PADDED_ATOMS*2], static_cast<unsigned long long>((long long) (forceJ.z*0x100000000)));
        } // end atomJ
        /*
        float3 origin = make_float3(1/((real) 0x100000000) * (long long) forceBuffers[atomI] ,
                1/((real) 0x100000000) * (long long)forceBuffers[atomI+NUM_PADDED_ATOMS],
                1/((real) 0x100000000) * (long long)forceBuffers[atomI+NUM_PADDED_ATOMS*2]);
                */
        if(abs(forceI.x)>1e-10) atomicAdd(&forceBuffers[atomI], static_cast<unsigned long long>((long long) (forceI.x*0x100000000)));
        if(abs(forceI.y)>1e-10) atomicAdd(&forceBuffers[atomI+NUM_PADDED_ATOMS], static_cast<unsigned long long>((long long) (forceI.y*0x100000000)));
        if(abs(forceI.z)>1e-10) atomicAdd(&forceBuffers[atomI+NUM_PADDED_ATOMS*2], static_cast<unsigned long long>((long long) (forceI.z*0x100000000)));
        /*
        float3 end = make_float3(1/((real) 0x100000000) * (long long) forceBuffers[atomI] ,
                1/((real) 0x100000000) * (long long)forceBuffers[atomI+NUM_PADDED_ATOMS],
                1/((real) 0x100000000) * (long long)forceBuffers[atomI+NUM_PADDED_ATOMS*2]);
        if(end.x*end.x+end.y*end.y+end.z*end.z > 1e9){
            printf("%e,%e,%e + %e,%e,%e -> %e,%e,%e\n",forceI.x,forceI.y,forceI.z,origin.x,origin.y,origin.z,end.x,end.y,end.z);
        }
            */
        } // end quadrature point
    } // end atomI
}
